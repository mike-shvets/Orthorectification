#include "hip/hip_runtime.h"
/*

We use a term *tile* to identify the rectangular submatrices of the image.
Not to be confused with the blocks of threads.

*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#define DSM_MAX_TILES_PER_BLOCK 500
#define DSM_MAX_TILES_PER_THREAD 500

// threads per block
#define TPB_1D 16
#define TPB (TPB_1D * TPB_1D)
// satellite pixels per thread
#define SAT_PPT_1D 2
#define SAT_PPT (SAT_PPT_1D * SAT_PPT_1D)
// satellite pixels per block
#define SAT_PPB_1D (SAT_PPT_1D * TPB_1D)
#define SAT_PPB (SAT_PPB_1D * SAT_PPB_1D)
// DSM pixels per thread
#define DSM_PPT_1D 1
#define DSM_PPT (DSM_PPT_1D * DSM_PPT_1D)
// DSM pixels per block
#define DSM_PPB_1D (DSM_PPT_1D * TPB_1D)
// #define DSM_PPB (DSM_PPB_1D * DSM_PPB_1D)

// this needs to be large negative number
#define DSM_IGNORE_VALUE -1E5
// extern const float DSM_IGNORE_VALUE;
#define EPS 1E-3

#define DTYPE float

__device__ bool d_rectanglesIntersect(DTYPE* bbox1, DTYPE* bbox2) {
    if (bbox2[0] > bbox1[2] ||
        bbox2[1] > bbox1[3] ||
        bbox1[0] > bbox2[2] ||
        bbox1[1] > bbox2[3]) { return false; }
    else { return true; }
}

__device__ DTYPE d_area(DTYPE x1, DTYPE y1,
            DTYPE x2, DTYPE y2,
            DTYPE x3, DTYPE y3) {
    return abs(x1 * (y2 - y3) + x2 * (y3 - y1) + x3 * (y1 - y2)) / 2;
}

__device__ DTYPE d_interpolate_three(DTYPE x, DTYPE y,
                        DTYPE x1, DTYPE y1, DTYPE v1,
                        DTYPE x2, DTYPE y2, DTYPE v2, 
                        DTYPE x3, DTYPE y3, DTYPE v3) {
    DTYPE denom = (y2 - y3) * (x1 - x3) + (x3 - x2) * (y1 - y3);
    DTYPE w1 = ((y2 - y3) * (x - x3) + (x3 - x2) * (y - y3)) / denom;
    DTYPE w2 = ((y3 - y1) * (x - x3) + (x1 - x3) * (y - y3)) / denom;
    DTYPE w3 = 1. - w1 - w2; 

    return (w1 * v1 + w2 * v2 + w3 * v3);
}

__device__ bool d_inside_triangle(DTYPE x, DTYPE y,
                     DTYPE x1, DTYPE y1, 
                     DTYPE x2, DTYPE y2, 
                     DTYPE x3, DTYPE y3) {
    DTYPE A = d_area(x1, y1, x2, y2, x3, y3);    
    DTYPE A1 = d_area(x, y, x1, y1, x2, y2);
    DTYPE A2 = d_area(x, y, x3, y3, x1, y1);
    DTYPE A3 = d_area(x, y, x2, y2, x3, y3);
    return (abs(A1 + A2 + A3 - A) < EPS);
}

__global__ void kernelComputePointsNum(DTYPE* pX, DTYPE* pY, DTYPE* pZ,
                                       int* dsmPixelCounts,
                                       int nfaces, int dsm_width,
                                       int sat_width, int sat_height) {
    int iface = blockIdx.x * blockDim.x + threadIdx.x;
    if (iface < nfaces) {
        int faces_per_row = 2 * (dsm_width - 1);
        int irow = iface / faces_per_row;
        int icol = (iface % faces_per_row) / 2;
        int idx = irow * dsm_width + icol;

        int idx1, idx2, idx3;
        if (iface % 2 == 0) {
            // **
            // *
            idx1 = idx;
            idx2 = idx + 1;
            idx3 = idx + dsm_width;
        } else {
            //  *
            // **
            idx1 = idx + 1;
            idx2 = idx + dsm_width;
            idx3 = idx + dsm_width + 1;
        }

        if (pZ[idx1] < DSM_IGNORE_VALUE + 1 ||
            pZ[idx2] < DSM_IGNORE_VALUE + 1 ||
            pZ[idx3] < DSM_IGNORE_VALUE + 1) { return; }
        
        float x1, y1, x2, y2, x3, y3;
        x1 = pX[idx1];
        y1 = pY[idx1];
        x2 = pX[idx2];
        y2 = pY[idx2];
        x3 = pX[idx3];
        y3 = pY[idx3];
        int ymin = static_cast<int>( ceilf(fminf(fminf(y1, y2), y3)) );
        int xmin = static_cast<int>( ceilf(fminf(fminf(x1, x2), x3)) );
        int ymax = static_cast<int>( floorf(fmaxf(fmaxf(y1, y2), y3)) );
        int xmax = static_cast<int>( floorf(fmaxf(fmaxf(x1, x2), x3)) );

        ymin = fmaxf(0, ymin);
        xmin = fmaxf(0, xmin);
        ymax = fminf(sat_height - 1, ymax);
        xmax = fminf(sat_width - 1, xmax);

        //if ((xmax - xmin) * (ymax - ymin) > 100) {
        //    dsmPixelCounts[iface] = -1;
        //} else {
        {
            for (int x = xmin; x <= xmax; ++x) {
                for (int y = ymin; y <= ymax; ++y) {
                    if (d_inside_triangle((float) x - x1, (float) y - y1,
                                          0, 0, x2-x1, y2-y1, x3-x1, y3-y1)) {
                        dsmPixelCounts[iface] += 1;
                    }
                }
            }
        }
    }
}

__global__ void kernelGetPoints(DTYPE* pX, DTYPE* pY, DTYPE* pZ,
                                int* dsmPixelCounts,
                                int* faceIDs, int* pixelIDs,
                                int nfaces, int dsm_width,
                                int sat_width, int sat_height) {
    int iface = blockIdx.x * blockDim.x + threadIdx.x;
    if (iface < nfaces) {
        int curIdx = dsmPixelCounts[iface];

        int faces_per_row = 2 * (dsm_width - 1);
        int irow = iface / faces_per_row;
        int icol = (iface % faces_per_row) / 2;
        int idx = irow * dsm_width + icol;

        int idx1, idx2, idx3;
        if (iface % 2 == 0) {
            // **
            // *
            idx1 = idx;
            idx2 = idx + 1;
            idx3 = idx + dsm_width;
        } else {
            //  *
            // **
            idx1 = idx + 1;
            idx2 = idx + dsm_width;
            idx3 = idx + dsm_width + 1;
        }

        if (pZ[idx1] < DSM_IGNORE_VALUE + 1 ||
            pZ[idx2] < DSM_IGNORE_VALUE + 1 ||
            pZ[idx3] < DSM_IGNORE_VALUE + 1) { return; }
        
        float x1, y1, x2, y2, x3, y3;
        x1 = pX[idx1];
        y1 = pY[idx1];
        x2 = pX[idx2];
        y2 = pY[idx2];
        x3 = pX[idx3];
        y3 = pY[idx3];
        int ymin = static_cast<int>( ceilf(fminf(fminf(y1, y2), y3)) );
        int xmin = static_cast<int>( ceilf(fminf(fminf(x1, x2), x3)) );
        int ymax = static_cast<int>( floorf(fmaxf(fmaxf(y1, y2), y3)) );
        int xmax = static_cast<int>( floorf(fmaxf(fmaxf(x1, x2), x3)) );

        ymin = fmaxf(0, ymin);
        xmin = fmaxf(0, xmin);
        ymax = fminf(sat_height - 1, ymax);
        xmax = fminf(sat_width - 1, xmax);

        //if ((xmax - xmin) * (ymax - ymin) > 100) {
        //    dsmPixelCounts[iface] = -1;
        //} else {
        {
            for (int x = xmin; x <= xmax; ++x) {
                for (int y = ymin; y <= ymax; ++y) {
                    if (d_inside_triangle((float) x - x1, (float) y - y1,
                                          0, 0, x2-x1, y2-y1, x3-x1, y3-y1)) {
                        faceIDs[curIdx] = iface;
                        pixelIDs[curIdx] = y * sat_width + x;
                        curIdx++;
                    }
                }
            }
        }
    }
}

__global__ void kernelFindLimits(int* ids, int* limits, int num) {
    int iel = blockIdx.x * blockDim.x + threadIdx.x;
    if (iel < num) {
        int pixelID = ids[iel];
        if (iel == 0 || ids[iel - 1] != pixelID) {
            limits[pixelID * 2 + 0] = iel;
        }
        if (iel == num - 1 || ids[iel + 1] != pixelID) {
            limits[pixelID * 2 + 1] = iel + 1;
        }
    }
}

__global__ void kernelDraw(int* faceIDs, int* pixelIDsLimits,
                           float* pX, float* pY, float* pZ,
                           float* pOut,
                           int sat_npixels, int dsm_width, int sat_width) {
    int ipixel = blockIdx.x * blockDim.x + threadIdx.x;
    if (ipixel < sat_npixels) {
        int faces_per_row = 2 * (dsm_width - 1);
        for (int i = pixelIDsLimits[2 * ipixel + 0];
                 i < pixelIDsLimits[2 * ipixel + 1]; ++i) {
            int iface = faceIDs[i];

            int irow = iface / faces_per_row;
            int icol = (iface % faces_per_row) / 2;
            int idx = irow * dsm_width + icol;

            int idx1, idx2, idx3;
            if (iface % 2 == 0) {
                // **
                // *
                idx1 = idx;
                idx2 = idx + 1;
                idx3 = idx + dsm_width;
            } else {
                //  *
                // **
                idx1 = idx + 1;
                idx2 = idx + dsm_width;
                idx3 = idx + dsm_width + 1;
            }

            float x1, y1, elev1, x2, y2, elev2, x3, y3, elev3;
            x1 = pX[idx1];
            y1 = pY[idx1];
            elev1 = pZ[idx1];
            x2 = pX[idx2];
            y2 = pY[idx2];
            elev2 = pZ[idx2];
            x3 = pX[idx3];
            y3 = pY[idx3];
            elev3 = pZ[idx3];

            float x = static_cast<float>(ipixel % sat_width);
            float y = static_cast<float>(ipixel / sat_width);
            
            float elev = d_interpolate_three(x, y,
                                             x1, y1, elev1,
                                             x2, y2, elev2,
                                             x3, y3, elev3);

            if (elev > pOut[ipixel]) {
                pOut[ipixel] = elev;
            }
        }
    }
}

void cudaRenderSatElevation(DTYPE * pX, DTYPE* pY, DTYPE* pZ, DTYPE* pOut,
                            int dsm_width, int dsm_height, int sat_width, int sat_height) {
    int dsm_npixels = dsm_width * dsm_height;
    int sat_npixels = sat_width * sat_height;
    
    DTYPE* d_pX;
    DTYPE* d_pY;
    DTYPE* d_pZ;
    DTYPE* d_pOut;
    hipMalloc((void **)&d_pX, sizeof(DTYPE) * dsm_npixels);
    hipMalloc((void **)&d_pY, sizeof(DTYPE) * dsm_npixels);
    hipMalloc((void **)&d_pZ, sizeof(DTYPE) * dsm_npixels);
    hipMalloc((void **)&d_pOut, sizeof(DTYPE) * sat_npixels);
    hipMemcpy(d_pX, pX, sizeof(DTYPE) * dsm_npixels, hipMemcpyHostToDevice);
    hipMemcpy(d_pY, pY, sizeof(DTYPE) * dsm_npixels, hipMemcpyHostToDevice);
    hipMemcpy(d_pZ, pZ, sizeof(DTYPE) * dsm_npixels, hipMemcpyHostToDevice);
    // output memory on host contains all min values
    hipMemcpy(d_pOut, pOut, sizeof(DTYPE) * sat_npixels, hipMemcpyHostToDevice);

    int nfaces = 2 * (dsm_height - 1) * (dsm_width - 1);
    int nblocks = (nfaces + TPB - 1) / TPB;
    // compute # of pixels that each face cover
    // TODO: change to int
    int* dsmPixelCounts;
    hipMalloc((void **)&dsmPixelCounts, sizeof(int) * nfaces);
    hipMemset(dsmPixelCounts, 0, sizeof(int) * nfaces);
    kernelComputePointsNum<<<nblocks, TPB>>>(d_pX, d_pY, d_pZ,
                                             dsmPixelCounts, nfaces,
                                             dsm_width, sat_width, sat_height);
    // hipDeviceSynchronize();
    hipDeviceSynchronize();
    if ( hipSuccess != hipGetLastError() )
        printf( "Error in CUDA kernel attempting to compute number of points "
                "for each thread!\n" );

    int numPixelsLast;
    hipMemcpy(&numPixelsLast, dsmPixelCounts + nfaces - 1, sizeof(int),
               hipMemcpyDeviceToHost);

    // exclusive scan to get start index for each face
    thrust::exclusive_scan(thrust::device, dsmPixelCounts,
                           dsmPixelCounts + nfaces, dsmPixelCounts);

    //
    int numPixelsTotal;
    hipMemcpy(&numPixelsTotal, dsmPixelCounts + nfaces - 1, sizeof(int),
               hipMemcpyDeviceToHost);
    numPixelsTotal += numPixelsLast;
    printf("================= %d\n", numPixelsTotal);
    int* faceIDs;
    int* pixelIDs;
    hipMalloc((void **)&faceIDs, sizeof(int) * numPixelsTotal);
    hipMalloc((void **)&pixelIDs, sizeof(int) * numPixelsTotal);
    kernelGetPoints<<<nblocks, TPB>>>(d_pX, d_pY, d_pZ,
                                      dsmPixelCounts,
                                      faceIDs, pixelIDs,
                                      nfaces,
                                      dsm_width, sat_width, sat_height);
    hipDeviceSynchronize();
    if ( hipSuccess != hipGetLastError() )
        printf( "Error in CUDA kernel attempting to "
                "get points ids for each face!\n" );

    // sort by key
    thrust::sort_by_key(thrust::device, pixelIDs, pixelIDs + numPixelsTotal,
                        faceIDs);
    hipDeviceSynchronize();
    if ( hipSuccess != hipGetLastError() )
        printf( "Error in CUDA kernel attempting to "
                "sort!\n" );
   
    // find start and end points for each pixel
    int* pixelIDsLimits;
    hipMalloc((void **)&pixelIDsLimits, 2 * sizeof(int) * sat_npixels);
    hipMemset(pixelIDsLimits, 0, 2 * sizeof(int) * sat_npixels);
    nblocks = (numPixelsTotal + TPB - 1) / TPB;
    kernelFindLimits<<<nblocks, TPB>>>(pixelIDs, pixelIDsLimits, numPixelsTotal);
    hipDeviceSynchronize();
    if ( hipSuccess != hipGetLastError() )
        printf( "Error in CUDA kernel attempting to "
                "find start and end positions for each pixel!\n" );

    //
    nblocks = (sat_npixels + TPB - 1) / TPB;
    kernelDraw<<<nblocks, TPB>>>(faceIDs, pixelIDsLimits, d_pX, d_pY, d_pZ,
                                d_pOut,
                                sat_npixels, dsm_width, sat_width);
    hipDeviceSynchronize();
    if ( hipSuccess != hipGetLastError() )
        printf( "Error in CUDA kernel attempting to "
                "draw satellite elevation!\n" );

    
    // hipMemcpy(pOut, dsmPixelCounts, sizeof(float) * min(sat_npixels, nfaces), hipMemcpyDeviceToHost);
    hipMemcpy(pOut, d_pOut, sizeof(DTYPE) * sat_npixels,
               hipMemcpyDeviceToHost);
}

